#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "cu_queue.cuh"
#include "myQueue.cuh"

#include "Lock.cuh"

#include <stdio.h>


hipError_t cudaHelper();


__global__ void kernel(Lock* lck, cu_queue<int>* q) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	myQueue<int> m_q(lck, q);
	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	//int i = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if (i == 0) {
		int firstVal = 0;
		m_q.push(firstVal);
	}
	else{
		m_q.push(i);	
	}
	
	__syncthreads();
}

int main()
{
	hipError_t cudaStatus = cudaHelper();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t cudaHelper() {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cu_queue<int>* h_q = new cu_queue<int>();	//host queue
	cu_queue<int>* d_q;							//device queue
	Lock* h_lock = new Lock();
	Lock* d_lock;

	int* tmp_mutex; //to hold h_lock->mutex


	cudaStatus = hipMalloc((void**)&d_q, sizeof(cu_queue<int>));
	cudaStatus = hipMalloc((void**)&d_lock, sizeof(Lock));

	cudaStatus = hipMalloc((void**)&tmp_mutex, sizeof(int));
	//cudaStatus = hipMalloc((void**)&d_lock->mutex, 1 * sizeof(int)); //does not work

	/*IN GENERAL, members of a class must be copied manually into Device instances*/

	//Tricky trick to copy memory of a pointer within an pointer instance of a class
	cudaStatus = hipMemcpy(tmp_mutex, h_lock->mutex, sizeof(int), hipMemcpyHostToDevice); \

	cudaStatus = hipMemcpy(d_q, h_q, sizeof(cu_queue<int>), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_lock, h_lock, sizeof(Lock), hipMemcpyHostToDevice);

	//Tricky trick to copy memory of a pointer within a pointer instance of a class
	cudaStatus = hipMemcpy(&(d_lock->mutex), &tmp_mutex, sizeof(int*), hipMemcpyHostToDevice);


	dim3 gridSize(32, 1, 1);	//Number of blocks
	dim3 blockSize(1, 1, 1);	//Number of threads per block, max=1024, depending on GPU

	kernel << <gridSize, blockSize >> > (d_lock, d_q);

	cudaStatus = hipGetLastError();
	cudaStatus = hipDeviceSynchronize();

	return cudaStatus;
}